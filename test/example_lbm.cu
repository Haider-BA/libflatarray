#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <libflatarray/flat_array.hpp>
#include <boost/date_time/posix_time/posix_time.hpp>
#include <stdexcept>

long long time_usec()
{
    boost::posix_time::ptime now = boost::posix_time::microsec_clock::local_time();
    return now.time_of_day().total_microseconds();
}

void check_cuda_error()
{
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(error) << "\n";
        throw std::runtime_error("CUDA error");
    }
}

class CellLBM
{
public:
    double C;
    double N;
    double E;
    double W;
    double S;
    double T;
    double B;
    double NW;
    double NE;
    double SW;
    double SE;
    double TW;
    double BW;
    double TE;
    double BE;
    double TN;
    double BN;
    double TS;
    double BS;

#define GET_COMP(X, Y, Z, DIR)                                  \
    accessorOld[LibFlatArray::FixedCoord<X, Y, Z>()].DIR()

#define SET_COMP(DIR)                           \
    accessorNew.DIR()

    // fixme: refactor interface so that all wire-up in Cell can be summarized?
    template<typename ACCESSOR1, typename ACCESSOR2>
    __device__
    __host__
    static void updateLine(ACCESSOR1 accessorOld, int *indexOld, ACCESSOR2 accessorNew, int *indexNew, int startZ, int endZ)
    {
        int global_x = blockIdx.x * blockDim.x + threadIdx.x + 2;
        int global_y = blockIdx.y * blockDim.y + threadIdx.y + 2;
        int global_z = startZ;
        *indexOld =
            global_z * ACCESSOR1::DIM_X * ACCESSOR1::DIM_Y +
            global_y * ACCESSOR1::DIM_X +
            global_x;
        *indexNew =
            global_z * ACCESSOR2::DIM_X * ACCESSOR2::DIM_Y +
            global_y * ACCESSOR2::DIM_X +
            global_x;
        const int planeSizeOld = ACCESSOR1::DIM_X * ACCESSOR1::DIM_Y;
        const int planeSizeNew = ACCESSOR1::DIM_X * ACCESSOR2::DIM_Y;

#pragma unroll 10
        for (; global_z < endZ; global_z += 1) {
// #define SQR(X) ((X)*(X))
//             const double omega = 1.0/1.7;
//             const double omega_trm = 1.0 - omega;
//             const double omega_w0 = 3.0 * 1.0 / 3.0 * omega;
//             const double omega_w1 = 3.0*1.0/18.0*omega;
//             const double omega_w2 = 3.0*1.0/36.0*omega;
//             const double one_third = 1.0 / 3.0;
//             double velX, velY, velZ;

//             velX  =
//                 GET_COMP(-1,0,0,E) + GET_COMP(x-1,y-1,0,NE) +
//                 GET_COMP(-1,1,0,SE) + GET_COMP(x-1,y,z-1,TE) +
//                 GET_COMP(-1,0,1,BE);
//             velY  = GET_COMP(x,y-1,0,N) + GET_COMP(x+1,y-1,0,NW) +
//                 GET_COMP(x,y-1,z-1,TN) + GET_COMP(x,y-1,1,BN);
//             velZ  = GET_COMP(x,y,z-1,T) + GET_COMP(x,y+1,z-1,TS) +
//                 GET_COMP(x+1,y,z-1,TW);

//             const double rho =
//                 GET_COMP(x,y,0,C) + GET_COMP(x,y+1,0,S) +
//                 GET_COMP(x+1,y,0,W) + GET_COMP(x,y,1,B) +
//                 GET_COMP(x+1,y+1,0,SW) + GET_COMP(x,y+1,1,BS) +
//                 GET_COMP(x+1,y,1,BW) + velX + velY + velZ;
//             velX  = velX
//                 - GET_COMP(x+1,y,0,W)    - GET_COMP(x+1,y-1,0,NW)
//                 - GET_COMP(x+1,y+1,0,SW) - GET_COMP(x+1,y,z-1,TW)
//                 - GET_COMP(x+1,y,1,BW);
//             velY  = velY
//                 + GET_COMP(x-1,y-1,0,NE) - GET_COMP(x,y+1,0,S)
//                 - GET_COMP(x+1,y+1,0,SW) - GET_COMP(x-1,y+1,0,SE)
//                 - GET_COMP(x,y+1,z-1,TS) - GET_COMP(x,y+1,1,BS);
//             velZ  = velGET_COMP(x,y-1,z-1,TN) + GET_COMP(x-1,y,z-1,TE) - GET_COMP(x,y,1,B) - GET_COMP(x,y-1,1,BN) - GET_COMP(x,y+1,1,BS) - GET_COMP(x+1,y,1,BW) - GET_COMP(x-1,y,1,BE);

//             // density = rho;
//             // velocityX = velX;
//             // velocityY = velY;
//             // velocityZ = velZ;

//             const double dir_indep_trm = one_third*rho - 0.5*( velX*velX + velY*velY + velZ*velZ );

//             SET_COMP(C)=omega_trm * GET_COMP(x,y,0,C) + omega_w0*( dir_indep_trm );

//             SET_COMP(NW)=omega_trm * GET_COMP(x+1,y-1,0,NW) +
//                 omega_w2*( dir_indep_trm - ( velX-velY ) + 1.5*SQR( velX-velY ) );
//             SET_COMP(SE)=omega_trm * GET_COMP(x-1,y+1,0,SE) +
//                 omega_w2*( dir_indep_trm + ( velX-velY ) + 1.5*SQR( velX-velY ) );
//             SET_COMP(NE)=omega_trm * GET_COMP(x-1,y-1,0,NE) +
//                 omega_w2*( dir_indep_trm + ( velX+velY ) + 1.5*SQR( velX+velY ) );
//             SET_COMP(SW)=omega_trm * GET_COMP(x+1,y+1,0,SW) +
//                 omega_w2*( dir_indep_trm - ( velX+velY ) + 1.5*SQR( velX+velY ) );

//             SET_COMP(TW)=omega_trm * GET_COMP(x+1,y,z-1,TW) + omega_w2*( dir_indep_trm - ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
//             SET_COMP(BE)=omega_trm * GET_COMP(x-1,y,1,BE) + omega_w2*( dir_indep_trm + ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
//             SET_COMP(TE)=omega_trm * GET_COMP(x-1,y,z-1,TE) + omega_w2*( dir_indep_trm + ( velX+velZ ) + 1.5*SQR( velX+velZ ) );
//             SET_COMP(BW)=omega_trm * GET_COMP(x+1,y,1,BW) + omega_w2*( dir_indep_trm - ( velX+velZ ) + 1.5*SQR( velX+velZ ) );

//             SET_COMP(TS)=omega_trm * GET_COMP(x,y+1,z-1,TS) + omega_w2*( dir_indep_trm - ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
//             SET_COMP(BN)=omega_trm * GET_COMP(x,y-1,1,BN) + omega_w2*( dir_indep_trm + ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
//             SET_COMP(TN)=omega_trm * GET_COMP(x,y-1,z-1,TN) + omega_w2*( dir_indep_trm + ( velY+velZ ) + 1.5*SQR( velY+velZ ) );
//             SET_COMP(BS)=omega_trm * GET_COMP(x,y+1,1,BS) + omega_w2*( dir_indep_trm - ( velY+velZ ) + 1.5*SQR( velY+velZ ) );

//             SET_COMP(N)=omega_trm * GET_COMP(x,y-1,0,N) + omega_w1*( dir_indep_trm + velY + 1.5*SQR(velY));
//             SET_COMP(S)=omega_trm * GET_COMP(x,y+1,0,S) + omega_w1*( dir_indep_trm - velY + 1.5*SQR(velY));
//             SET_COMP(E)=omega_trm * GET_COMP(x-1,y,0,E) + omega_w1*( dir_indep_trm + velX + 1.5*SQR(velX));
//             SET_COMP(W)=omega_trm * GET_COMP(x+1,y,0,W) + omega_w1*( dir_indep_trm - velX + 1.5*SQR(velX));
//             SET_COMP(T)=omega_trm * GET_COMP(x,y,z-1,T) + omega_w1*( dir_indep_trm + velZ + 1.5*SQR(velZ));
//             SET_COMP(B)=omega_trm * GET_COMP(x,y,1,B) + omega_w1*( dir_indep_trm - velZ + 1.5*SQR(velZ));


            SET_COMP(C) = GET_COMP(0, 0, 0, C);
            // if ((x == 2) && (y == 2)) {
            //     printf("going strong: %d, %d, %d\n", z, planeSizeOld, planeSizeNew);
            // }
            *indexOld += planeSizeOld;
            *indexNew += planeSizeNew;
        }
    }

#undef GET_COMP
#undef SET_COMP
#undef SQR

};

LIBFLATARRAY_REGISTER_SOA(CellLBM, ((double)(C))((double)(N))((double)(E))((double)(W))((double)(S))((double)(T))((double)(B))((double)(NW))((double)(SW))((double)(NE))((double)(SE))((double)(TW))((double)(BW))((double)(TE))((double)(BE))((double)(TN))((double)(BN))((double)(TS))((double)(BS)))

#define C 0
#define N 1
#define E 2
#define W 3
#define S 4
#define T 5
#define B 6

#define NW 7
#define SW 8
#define NE 9
#define SE 10

#define TW 11
#define BW 12
#define TE 13
#define BE 14

#define TN 15
#define BN 16
#define TS 17
#define BS 18

#define GET_COMP(X, Y, Z, DIR)                                          \
    gridOld[(Z) * dimX * dimY + (Y) * dimX + (X) + (DIR) * dimX * dimY * dimZ]

#define SET_COMP(DIR)                                                   \
    gridNew[z   * dimX * dimY +   y * dimX +   x + (DIR) * dimX * dimY * dimZ]

template<int UNUSED_X, int UNUSED_Y, int UNUSED_Z>
__global__ void update_lbm_classic(int dimX, int dimY, int dimZ, double *gridOld, double *gridNew)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 2;
    int z = 2;

#pragma unroll 10
    for (; z < (dimZ - 2); z += 1) {

#define SQR(X) ((X)*(X))
        const double omega = 1.0/1.7;
        const double omega_trm = 1.0 - omega;
        const double omega_w0 = 3.0 * 1.0 / 3.0 * omega;
        const double omega_w1 = 3.0*1.0/18.0*omega;
        const double omega_w2 = 3.0*1.0/36.0*omega;
        const double one_third = 1.0 / 3.0;
        double velX, velY, velZ;

        velX  =
            GET_COMP(x-1,y,z,E) + GET_COMP(x-1,y-1,z,NE) +
            GET_COMP(x-1,y+1,z,SE) + GET_COMP(x-1,y,z-1,TE) +
            GET_COMP(x-1,y,z+1,BE);
        velY  = GET_COMP(x,y-1,z,N) + GET_COMP(x+1,y-1,z,NW) +
            GET_COMP(x,y-1,z-1,TN) + GET_COMP(x,y-1,z+1,BN);
        velZ  = GET_COMP(x,y,z-1,T) + GET_COMP(x,y+1,z-1,TS) +
            GET_COMP(x+1,y,z-1,TW);

        const double rho =
            GET_COMP(x,y,z,C) + GET_COMP(x,y+1,z,S) +
            GET_COMP(x+1,y,z,W) + GET_COMP(x,y,z+1,B) +
            GET_COMP(x+1,y+1,z,SW) + GET_COMP(x,y+1,z+1,BS) +
            GET_COMP(x+1,y,z+1,BW) + velX + velY + velZ;
        velX  = velX
            - GET_COMP(x+1,y,z,W)    - GET_COMP(x+1,y-1,z,NW)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x+1,y,z-1,TW)
            - GET_COMP(x+1,y,z+1,BW);
        velY  = velY
            + GET_COMP(x-1,y-1,z,NE) - GET_COMP(x,y+1,z,S)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x-1,y+1,z,SE)
            - GET_COMP(x,y+1,z-1,TS) - GET_COMP(x,y+1,z+1,BS);
        velZ  = velZ+GET_COMP(x,y-1,z-1,TN) + GET_COMP(x-1,y,z-1,TE) - GET_COMP(x,y,z+1,B) - GET_COMP(x,y-1,z+1,BN) - GET_COMP(x,y+1,z+1,BS) - GET_COMP(x+1,y,z+1,BW) - GET_COMP(x-1,y,z+1,BE);

        // density = rho;
        // velocityX = velX;
        // velocityY = velY;
        // velocityZ = velZ;

        const double dir_indep_trm = one_third*rho - 0.5*( velX*velX + velY*velY + velZ*velZ );

        SET_COMP(C)=omega_trm * GET_COMP(x,y,z,C) + omega_w0*( dir_indep_trm );

        SET_COMP(NW)=omega_trm * GET_COMP(x+1,y-1,z,NW) +
            omega_w2*( dir_indep_trm - ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(SE)=omega_trm * GET_COMP(x-1,y+1,z,SE) +
            omega_w2*( dir_indep_trm + ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(NE)=omega_trm * GET_COMP(x-1,y-1,z,NE) +
            omega_w2*( dir_indep_trm + ( velX+velY ) + 1.5*SQR( velX+velY ) );
        SET_COMP(SW)=omega_trm * GET_COMP(x+1,y+1,z,SW) +
            omega_w2*( dir_indep_trm - ( velX+velY ) + 1.5*SQR( velX+velY ) );

        SET_COMP(TW)=omega_trm * GET_COMP(x+1,y,z-1,TW) + omega_w2*( dir_indep_trm - ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(BE)=omega_trm * GET_COMP(x-1,y,z+1,BE) + omega_w2*( dir_indep_trm + ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(TE)=omega_trm * GET_COMP(x-1,y,z-1,TE) + omega_w2*( dir_indep_trm + ( velX+velZ ) + 1.5*SQR( velX+velZ ) );
        SET_COMP(BW)=omega_trm * GET_COMP(x+1,y,z+1,BW) + omega_w2*( dir_indep_trm - ( velX+velZ ) + 1.5*SQR( velX+velZ ) );

        SET_COMP(TS)=omega_trm * GET_COMP(x,y+1,z-1,TS) + omega_w2*( dir_indep_trm - ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(BN)=omega_trm * GET_COMP(x,y-1,z+1,BN) + omega_w2*( dir_indep_trm + ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(TN)=omega_trm * GET_COMP(x,y-1,z-1,TN) + omega_w2*( dir_indep_trm + ( velY+velZ ) + 1.5*SQR( velY+velZ ) );
        SET_COMP(BS)=omega_trm * GET_COMP(x,y+1,z+1,BS) + omega_w2*( dir_indep_trm - ( velY+velZ ) + 1.5*SQR( velY+velZ ) );

        SET_COMP(N)=omega_trm * GET_COMP(x,y-1,z,N) + omega_w1*( dir_indep_trm + velY + 1.5*SQR(velY));
        SET_COMP(S)=omega_trm * GET_COMP(x,y+1,z,S) + omega_w1*( dir_indep_trm - velY + 1.5*SQR(velY));
        SET_COMP(E)=omega_trm * GET_COMP(x-1,y,z,E) + omega_w1*( dir_indep_trm + velX + 1.5*SQR(velX));
        SET_COMP(W)=omega_trm * GET_COMP(x+1,y,z,W) + omega_w1*( dir_indep_trm - velX + 1.5*SQR(velX));
        SET_COMP(T)=omega_trm * GET_COMP(x,y,z-1,T) + omega_w1*( dir_indep_trm + velZ + 1.5*SQR(velZ));
        SET_COMP(B)=omega_trm * GET_COMP(x,y,z+1,B) + omega_w1*( dir_indep_trm - velZ + 1.5*SQR(velZ));
    }
}

#undef GET_COMP
#undef SET_COMP
#undef SQR

#undef C
#undef N
#undef E
#undef W
#undef S
#undef T
#undef B

#undef NW
#undef SW
#undef NE
#undef SE

#undef TW
#undef BW
#undef TE
#undef BE

#undef TN
#undef BN
#undef TS
#undef BS

#define GET_COMP(X, Y, Z, DIR)                  \
    hoodOld[LibFlatArray::FixedCoord<X, Y, Z>()].DIR()

#define SET_COMP(DIR)                           \
    hoodNew.DIR()

template<int DIM_X, int DIM_Y, int DIM_Z>
__global__ void update_lbm_flat_array(int dimX, int dimY, int dimZ, double *gridOld, double *gridNew)
{
    int myX = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int myY = blockIdx.y * blockDim.y + threadIdx.y + 2;
    int myZ = 2;

    int index = myZ * DIM_X * DIM_Y + myY * DIM_X + myX;
    int offset = DIM_X * DIM_Y;
    int end = DIM_X * DIM_Y * (dimZ - 2);

    LibFlatArray::soa_accessor<CellLBM, DIM_X, DIM_Y, DIM_Z, 0> hoodNew((char*)gridNew, &index);
    LibFlatArray::soa_accessor<CellLBM, DIM_X, DIM_Y, DIM_Z, 0> hoodOld((char*)gridOld, &index);

#pragma unroll 10
    for (; index < end; index += offset) {
#define SQR(X) ((X)*(X))
        const double omega = 1.0/1.7;
        const double omega_trm = 1.0 - omega;
        const double omega_w0 = 3.0 * 1.0 / 3.0 * omega;
        const double omega_w1 = 3.0*1.0/18.0*omega;
        const double omega_w2 = 3.0*1.0/36.0*omega;
        const double one_third = 1.0 / 3.0;
        const int x = 0;
        const int y = 0;
        const int z = 0;
        double velX, velY, velZ;

        velX  =
            GET_COMP(x-1,y,z,E) + GET_COMP(x-1,y-1,z,NE) +
            GET_COMP(x-1,y+1,z,SE) + GET_COMP(x-1,y,z-1,TE) +
            GET_COMP(x-1,y,z+1,BE);
        velY  = GET_COMP(x,y-1,z,N) + GET_COMP(x+1,y-1,z,NW) +
            GET_COMP(x,y-1,z-1,TN) + GET_COMP(x,y-1,z+1,BN);
        velZ  = GET_COMP(x,y,z-1,T) + GET_COMP(x,y+1,z-1,TS) +
            GET_COMP(x+1,y,z-1,TW);

        const double rho =
            GET_COMP(x,y,z,C) + GET_COMP(x,y+1,z,S) +
            GET_COMP(x+1,y,z,W) + GET_COMP(x,y,z+1,B) +
            GET_COMP(x+1,y+1,z,SW) + GET_COMP(x,y+1,z+1,BS) +
            GET_COMP(x+1,y,z+1,BW) + velX + velY + velZ;
        velX  = velX
            - GET_COMP(x+1,y,z,W)    - GET_COMP(x+1,y-1,z,NW)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x+1,y,z-1,TW)
            - GET_COMP(x+1,y,z+1,BW);
        velY  = velY
            + GET_COMP(x-1,y-1,z,NE) - GET_COMP(x,y+1,z,S)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x-1,y+1,z,SE)
            - GET_COMP(x,y+1,z-1,TS) - GET_COMP(x,y+1,z+1,BS);
        velZ  = velZ+GET_COMP(x,y-1,z-1,TN) + GET_COMP(x-1,y,z-1,TE) - GET_COMP(x,y,z+1,B) - GET_COMP(x,y-1,z+1,BN) - GET_COMP(x,y+1,z+1,BS) - GET_COMP(x+1,y,z+1,BW) - GET_COMP(x-1,y,z+1,BE);

        // density = rho;
        // velocityX = velX;
        // velocityY = velY;
        // velocityZ = velZ;

        const double dir_indep_trm = one_third*rho - 0.5*( velX*velX + velY*velY + velZ*velZ );

        SET_COMP(C)=omega_trm * GET_COMP(x,y,z,C) + omega_w0*( dir_indep_trm );

        SET_COMP(NW)=omega_trm * GET_COMP(x+1,y-1,z,NW) +
            omega_w2*( dir_indep_trm - ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(SE)=omega_trm * GET_COMP(x-1,y+1,z,SE) +
            omega_w2*( dir_indep_trm + ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(NE)=omega_trm * GET_COMP(x-1,y-1,z,NE) +
            omega_w2*( dir_indep_trm + ( velX+velY ) + 1.5*SQR( velX+velY ) );
        SET_COMP(SW)=omega_trm * GET_COMP(x+1,y+1,z,SW) +
            omega_w2*( dir_indep_trm - ( velX+velY ) + 1.5*SQR( velX+velY ) );

        SET_COMP(TW)=omega_trm * GET_COMP(x+1,y,z-1,TW) + omega_w2*( dir_indep_trm - ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(BE)=omega_trm * GET_COMP(x-1,y,z+1,BE) + omega_w2*( dir_indep_trm + ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(TE)=omega_trm * GET_COMP(x-1,y,z-1,TE) + omega_w2*( dir_indep_trm + ( velX+velZ ) + 1.5*SQR( velX+velZ ) );
        SET_COMP(BW)=omega_trm * GET_COMP(x+1,y,z+1,BW) + omega_w2*( dir_indep_trm - ( velX+velZ ) + 1.5*SQR( velX+velZ ) );

        SET_COMP(TS)=omega_trm * GET_COMP(x,y+1,z-1,TS) + omega_w2*( dir_indep_trm - ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(BN)=omega_trm * GET_COMP(x,y-1,z+1,BN) + omega_w2*( dir_indep_trm + ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(TN)=omega_trm * GET_COMP(x,y-1,z-1,TN) + omega_w2*( dir_indep_trm + ( velY+velZ ) + 1.5*SQR( velY+velZ ) );
        SET_COMP(BS)=omega_trm * GET_COMP(x,y+1,z+1,BS) + omega_w2*( dir_indep_trm - ( velY+velZ ) + 1.5*SQR( velY+velZ ) );

        SET_COMP(N)=omega_trm * GET_COMP(x,y-1,z,N) + omega_w1*( dir_indep_trm + velY + 1.5*SQR(velY));
        SET_COMP(S)=omega_trm * GET_COMP(x,y+1,z,S) + omega_w1*( dir_indep_trm - velY + 1.5*SQR(velY));
        SET_COMP(E)=omega_trm * GET_COMP(x-1,y,z,E) + omega_w1*( dir_indep_trm + velX + 1.5*SQR(velX));
        SET_COMP(W)=omega_trm * GET_COMP(x+1,y,z,W) + omega_w1*( dir_indep_trm - velX + 1.5*SQR(velX));
        SET_COMP(T)=omega_trm * GET_COMP(x,y,z-1,T) + omega_w1*( dir_indep_trm + velZ + 1.5*SQR(velZ));
        SET_COMP(B)=omega_trm * GET_COMP(x,y,z+1,B) + omega_w1*( dir_indep_trm - velZ + 1.5*SQR(velZ));
    }
}

#undef GET_COMP
#undef SET_COMP

#define GET_COMP(X, Y, Z, DIR)                  \
    gridOld[(Z) * dimX * dimY + (Y) * dimX + (X)].DIR

#define SET_COMP(DIR)                           \
    gridNew[(z) * dimX * dimY + (y) * dimX + (x)].DIR

__global__ void update_lbm_object_oriented(int dimX, int dimY, int dimZ, CellLBM *gridOld, CellLBM *gridNew)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int y = blockIdx.y * blockDim.y + threadIdx.y + 2;
    int z = 2;

#pragma unroll 10
    for (; z < (dimZ - 2); z += 1) {

#define SQR(X) ((X)*(X))
        const double omega = 1.0/1.7;
        const double omega_trm = 1.0 - omega;
        const double omega_w0 = 3.0 * 1.0 / 3.0 * omega;
        const double omega_w1 = 3.0*1.0/18.0*omega;
        const double omega_w2 = 3.0*1.0/36.0*omega;
        const double one_third = 1.0 / 3.0;
        double velX, velY, velZ;

        velX  =
            GET_COMP(x-1,y,z,E) + GET_COMP(x-1,y-1,z,NE) +
            GET_COMP(x-1,y+1,z,SE) + GET_COMP(x-1,y,z-1,TE) +
            GET_COMP(x-1,y,z+1,BE);
        velY  = GET_COMP(x,y-1,z,N) + GET_COMP(x+1,y-1,z,NW) +
            GET_COMP(x,y-1,z-1,TN) + GET_COMP(x,y-1,z+1,BN);
        velZ  = GET_COMP(x,y,z-1,T) + GET_COMP(x,y+1,z-1,TS) +
            GET_COMP(x+1,y,z-1,TW);

        const double rho =
            GET_COMP(x,y,z,C) + GET_COMP(x,y+1,z,S) +
            GET_COMP(x+1,y,z,W) + GET_COMP(x,y,z+1,B) +
            GET_COMP(x+1,y+1,z,SW) + GET_COMP(x,y+1,z+1,BS) +
            GET_COMP(x+1,y,z+1,BW) + velX + velY + velZ;
        velX  = velX
            - GET_COMP(x+1,y,z,W)    - GET_COMP(x+1,y-1,z,NW)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x+1,y,z-1,TW)
            - GET_COMP(x+1,y,z+1,BW);
        velY  = velY
            + GET_COMP(x-1,y-1,z,NE) - GET_COMP(x,y+1,z,S)
            - GET_COMP(x+1,y+1,z,SW) - GET_COMP(x-1,y+1,z,SE)
            - GET_COMP(x,y+1,z-1,TS) - GET_COMP(x,y+1,z+1,BS);
        velZ  = velZ+GET_COMP(x,y-1,z-1,TN) + GET_COMP(x-1,y,z-1,TE) - GET_COMP(x,y,z+1,B) - GET_COMP(x,y-1,z+1,BN) - GET_COMP(x,y+1,z+1,BS) - GET_COMP(x+1,y,z+1,BW) - GET_COMP(x-1,y,z+1,BE);

        // density = rho;
        // velocityX = velX;
        // velocityY = velY;
        // velocityZ = velZ;

        const double dir_indep_trm = one_third*rho - 0.5*( velX*velX + velY*velY + velZ*velZ );

        SET_COMP(C)=omega_trm * GET_COMP(x,y,z,C) + omega_w0*( dir_indep_trm );

        SET_COMP(NW)=omega_trm * GET_COMP(x+1,y-1,z,NW) +
            omega_w2*( dir_indep_trm - ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(SE)=omega_trm * GET_COMP(x-1,y+1,z,SE) +
            omega_w2*( dir_indep_trm + ( velX-velY ) + 1.5*SQR( velX-velY ) );
        SET_COMP(NE)=omega_trm * GET_COMP(x-1,y-1,z,NE) +
            omega_w2*( dir_indep_trm + ( velX+velY ) + 1.5*SQR( velX+velY ) );
        SET_COMP(SW)=omega_trm * GET_COMP(x+1,y+1,z,SW) +
            omega_w2*( dir_indep_trm - ( velX+velY ) + 1.5*SQR( velX+velY ) );

        SET_COMP(TW)=omega_trm * GET_COMP(x+1,y,z-1,TW) + omega_w2*( dir_indep_trm - ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(BE)=omega_trm * GET_COMP(x-1,y,z+1,BE) + omega_w2*( dir_indep_trm + ( velX-velZ ) + 1.5*SQR( velX-velZ ) );
        SET_COMP(TE)=omega_trm * GET_COMP(x-1,y,z-1,TE) + omega_w2*( dir_indep_trm + ( velX+velZ ) + 1.5*SQR( velX+velZ ) );
        SET_COMP(BW)=omega_trm * GET_COMP(x+1,y,z+1,BW) + omega_w2*( dir_indep_trm - ( velX+velZ ) + 1.5*SQR( velX+velZ ) );

        SET_COMP(TS)=omega_trm * GET_COMP(x,y+1,z-1,TS) + omega_w2*( dir_indep_trm - ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(BN)=omega_trm * GET_COMP(x,y-1,z+1,BN) + omega_w2*( dir_indep_trm + ( velY-velZ ) + 1.5*SQR( velY-velZ ) );
        SET_COMP(TN)=omega_trm * GET_COMP(x,y-1,z-1,TN) + omega_w2*( dir_indep_trm + ( velY+velZ ) + 1.5*SQR( velY+velZ ) );
        SET_COMP(BS)=omega_trm * GET_COMP(x,y+1,z+1,BS) + omega_w2*( dir_indep_trm - ( velY+velZ ) + 1.5*SQR( velY+velZ ) );

        SET_COMP(N)=omega_trm * GET_COMP(x,y-1,z,N) + omega_w1*( dir_indep_trm + velY + 1.5*SQR(velY));
        SET_COMP(S)=omega_trm * GET_COMP(x,y+1,z,S) + omega_w1*( dir_indep_trm - velY + 1.5*SQR(velY));
        SET_COMP(E)=omega_trm * GET_COMP(x-1,y,z,E) + omega_w1*( dir_indep_trm + velX + 1.5*SQR(velX));
        SET_COMP(W)=omega_trm * GET_COMP(x+1,y,z,W) + omega_w1*( dir_indep_trm - velX + 1.5*SQR(velX));
        SET_COMP(T)=omega_trm * GET_COMP(x,y,z-1,T) + omega_w1*( dir_indep_trm + velZ + 1.5*SQR(velZ));
        SET_COMP(B)=omega_trm * GET_COMP(x,y,z+1,B) + omega_w1*( dir_indep_trm - velZ + 1.5*SQR(velZ));
    }
}

#undef GET_COMP
#undef SET_COMP

class benchmark
{
public:
    void evaluate()
    {
        for (int dim = 32; dim <= 160; dim += 4) {
            run(dim);
        }
    }

    void run(int dim)
    {
        int repeats = 100;
	if (dim <= 96) {
            repeats *= 10;
        }
        repeats = 1;

        long long useconds = exec(dim, repeats);

        double updates = 1.0 * gridSize(dim) * repeats;
        double seconds = useconds * 10e-6;
        double glups = 10e-9 * updates / seconds;

        std::cout << name() << " " << dim << " " << glups << " GLUPS\n";
    }

protected:
    virtual long long exec(int dim, int repeats) = 0;
    virtual std::string name() = 0;
    virtual size_t gridSize(int dim) = 0;
};

class benchmark_lbm_cuda : public benchmark
{
protected:
    long long exec(int dim, int repeats)
    {
        dim3 dimBlock;
        dim3 dimGrid;
        gen_dims(&dimBlock, &dimGrid, dim);

        return exec(dim, dimBlock, dimGrid, repeats);
    }

    virtual size_t gridSize(int dim)
    {
        dim3 dimBlock;
        dim3 dimGrid;
        gen_dims(&dimBlock, &dimGrid, dim);

        return dimGrid.x * dimBlock.x * dimGrid.y * dimBlock.y * (256 - 4);
    }

    virtual long long exec(int dim, dim3 dimBlock, dim3 dimGrid, int repeats) = 0;

    void gen_dims(dim3 *dimBlock, dim3 *dimGrid, int dim)
    {
        int blockWidth = 1;
        for (; blockWidth <= dim; blockWidth *= 2) {
        }
        blockWidth /= 2;
        blockWidth = std::min(256, blockWidth);
        *dimBlock = dim3(blockWidth, 2, 1);
        *dimGrid = dim3(dim / dimBlock->x, dim / dimBlock->y, 1);
    }
};


template<int DIM>
class benchmark_lbm_cuda_classic_callback
{
public:
    void operator()(int dim, long long *time, dim3 dimBlock, dim3 dimGrid, int repeats)
    {
        int size = (DIM + 2) * (DIM + 2) * (256 + 64) * 20;
        int bytesize = size * sizeof(double);
        std::vector<double> grid(size, 4711);

        double *devGridOld;
        double *devGridNew;
        hipMalloc(&devGridOld, bytesize);
        hipMalloc(&devGridNew, bytesize);
        check_cuda_error();

        hipMemcpy(devGridOld, &grid[0], bytesize, hipMemcpyHostToDevice);
        hipMemcpy(devGridNew, &grid[0], bytesize, hipMemcpyHostToDevice);
        check_cuda_error();

        hipDeviceSynchronize();
        long long t_start = time_usec();

        for (int t = 0; t < repeats; ++t) {
            update_lbm_flat_array<DIM, DIM, 256><<<dimGrid, dimBlock>>>(dim, dim, 256, devGridOld, devGridNew);
            // update_lbm_classic<DIM, DIM, 256><<<dimGrid, dimBlock>>>(dim, dim, 256, devGridOld, devGridNew);
            std::swap(devGridOld, devGridNew);
        }

        hipDeviceSynchronize();
        long long t_end = time_usec();
        check_cuda_error();

        hipMemcpy(&grid[0], devGridNew, bytesize, hipMemcpyDeviceToHost);
        hipFree(devGridOld);
        hipFree(devGridNew);
        check_cuda_error();
        *time = t_end - t_start;
    }
};

class benchmark_lbm_cuda_classic : public benchmark_lbm_cuda
{
protected:
    virtual long long exec(int dim, dim3 dimBlock, dim3 dimGrid, int repeats)
    {
        long long time;
        LibFlatArray::detail::flat_array::bind<benchmark_lbm_cuda_classic_callback>()(dim, &time, dimBlock, dimGrid, repeats);
        return time;
    }

    virtual std::string name()
    {
        return "lbm_cuda_classic";
    }
};

template<typename CELL, typename ACCESSOR1, typename ACCESSOR2>
__global__
void update(ACCESSOR1 accessor1, ACCESSOR2 accessor2)
{
    int indexOld;
    int indexNew;
    ACCESSOR1 accessorOld(accessor1.getData(), &indexOld);
    ACCESSOR2 accessorNew(accessor2.getData(), &indexNew);

    CELL::updateLine(accessorOld, &indexOld, accessorNew, &indexNew, 2, 256 - 2);
}

template<typename CELL, typename ACCESSOR1>
class SoAUpdateFunctorHelper2
{
public:
    SoAUpdateFunctorHelper2(ACCESSOR1 accessor1, int *index1, const dim3& dimBlock, const dim3& dimGrid) :
        accessor1(accessor1),
        index1(index1),
        dimBlock(dimBlock),
        dimGrid(dimGrid)
    {}

    template<typename ACCESSOR2>
    void operator()(ACCESSOR2 accessor2) const
    {
        // fixme: update this shit!
        // update<CELL><<<dimBlock, dimGrid>>>(accessor1, accessor2);
        update_lbm_flat_array<ACCESSOR1::DIM_X, ACCESSOR1::DIM_Y, ACCESSOR1::DIM_Z><<<dimBlock, dimGrid>>>(
            1, 1, 256, (double*)accessor1.get_data(), (double*)accessor2.get_data());
    }

    int index2;

private:
    ACCESSOR1 accessor1;
    int *index1;
    const dim3& dimBlock;
    const dim3& dimGrid;
};

template<typename CELL, typename GRID2>
class SoAUpdateFunctorHelper1
{
public:

    SoAUpdateFunctorHelper1(GRID2 *grid2, const dim3& dimBlock, const dim3& dimGrid) :
        grid2(grid2),
        dimBlock(dimBlock),
        dimGrid(dimGrid)
    {}

    template<typename ACCESSOR1>
    void operator()(ACCESSOR1 accessor1)
    {
        SoAUpdateFunctorHelper2<CELL, ACCESSOR1> helper(accessor1, &index1, dimBlock, dimGrid);
        grid2->callback(helper, &helper.index2);
    }

    int index1;

private:
    GRID2 *grid2;
    const dim3& dimBlock;
    const dim3& dimGrid;
};


template<typename CELL>
class SoAUpdateFunctorPrototype
{
public:
    template<typename GRID1, typename GRID2>
    void operator()(GRID1 *gridOld, GRID2 *gridNew, const dim3& dimBlock, const dim3& dimGrid)
    {
        SoAUpdateFunctorHelper1<CELL, GRID2> helper(gridNew, dimBlock, dimGrid);
        gridOld->callback(helper, &helper.index1);
    }
};

class benchmark_lbm_cuda_flat_array : public benchmark_lbm_cuda
{
    virtual long long exec(int dim, dim3 dimBlock, dim3 dimGrid, int repeats)
    {
        LibFlatArray::soa_grid<CellLBM> gridA(dim, dim, dim);
        LibFlatArray::soa_grid<CellLBM> gridB(dim, dim, dim);
        // fixme: init grid?

        char *dataA = gridA.get_data();
        char *dataB = gridB.get_data();

        char *buf;
        hipMalloc(reinterpret_cast<void**>(&buf), gridA.byte_size());
        gridA.set_data(buf);
        hipMalloc(reinterpret_cast<void**>(&buf), gridB.byte_size());
        gridB.set_data(buf);


        hipDeviceSynchronize();
        long long t_start = time_usec();

        LibFlatArray::soa_grid<CellLBM> *gridOld = &gridA;
        LibFlatArray::soa_grid<CellLBM> *gridNew = &gridB;

        // fixme: do the evolution. 1 functor per call?
        for (int t = 0; t < repeats; ++t) {
            SoAUpdateFunctorPrototype<CellLBM>()(gridOld, gridNew, dimBlock, dimGrid);
            std::swap(gridOld, gridNew);
        }

        hipDeviceSynchronize();
        long long t_end = time_usec();
        check_cuda_error();

        hipFree(gridA.get_data());
        hipFree(gridB.get_data());

        gridA.set_data(dataA);
        gridB.set_data(dataB);

        return t_end - t_start;
    }

    virtual std::string name()
    {
        return "lbm_cuda_flat_array";
    }
};

int main(int argc, char **argv)
{
    if (argc != 2) {
        std::cerr << "usage: " << argv[0] << " CUDA_DEVICE\n";
        return 1;
    }

    std::stringstream s;
    s << argv[1];
    int cudaDevice;
    s >> cudaDevice;
    hipSetDevice(cudaDevice);

    // benchmark_lbm_cuda_classic().evaluate();
    benchmark_lbm_cuda_flat_array().evaluate();

    return 0;
}
