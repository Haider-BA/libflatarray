/**
 * Copyright 2016 Andreas Schäfer
 *
 * Distributed under the Boost Software License, Version 1.0. (See accompanying
 * file LICENSE or copy at http://www.boost.org/LICENSE_1_0.txt)
 */

#include <boost/detail/lightweight_test.hpp>
#include <libflatarray/flat_array.hpp>
#include <map>

#include "test.hpp"

class ActiveElement
{
public:
    __host__
    __device__
    ActiveElement()
    {
        val += 100000;
    }

    __host__
    __device__
    ~ActiveElement()
    {
        val += 1000000;
    }

    inline bool operator==(ActiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class PassiveElement
{
public:
    inline bool operator==(PassiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class ConstructorDestructorTestCellActive
{
public:
    inline
    explicit ConstructorDestructorTestCellActive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellActive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellActive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    ActiveElement element;
};

class ConstructorDestructorTestCellPassive
{
public:
    inline
    explicit ConstructorDestructorTestCellPassive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellPassive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellPassive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    PassiveElement element;
};

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellActive,
                          ((double)(temperature))
                          ((ActiveElement)(element))
                          ((bool)(alive)) )

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellPassive,
                          ((double)(temperature))
                          ((PassiveElement)(element))
                          ((bool)(alive)) )

namespace LibFlatArray {

std::map<std::size_t, char*> allocation_cache;

/**
 * We fake allocation here to make sure our grids in the tests below
 * get the same pointers. We need this to be sure that we're working
 * on the same memory region with each.
 */
template<class T>
class fake_cuda_allocator
{
public:
    typedef ptrdiff_t difference_type;
    typedef T* pointer;
    typedef const T* const_pointer;
    typedef T& reference;
    typedef const T& const_reference;
    typedef T value_type;

    pointer allocate(std::size_t n, const void* = 0)
    {
        if (allocation_cache[n] != 0) {
            return allocation_cache[n];
        }


        pointer ret = 0;
        hipMalloc(&ret, n * sizeof(T));
        allocation_cache[n] = ret;
        return ret;
    }

    void deallocate(pointer p, std::size_t)
    {
        // intentionally left blank
    }

    void deallocate_all()
    {
        for (typename std::map<std::size_t, pointer>::iterator i = allocation_cache.begin(); i != allocation_cache.end(); ++i) {
            hipFree(i->second);
            i->second = 0;
        }
    }
};

ADD_TEST(TestCUDAConstructionDestruction)
{
    char *data = 0;
    {
        // prep device memory with consecutive numbers:
        soa_grid<ConstructorDestructorTestCellPassive, fake_cuda_allocator<char>, true> device_grid(20, 10, 5);
        data = device_grid.get_data();

        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell((x + 1) * (y + 1), true);
                    cell.element.val = x + y * 20 + z * 20 * 10;
                    host_grid.set(x, y, z, cell);

                    cell = host_grid.get(x, y, z);
                }
            }
        }
        hipMemcpy(device_grid.get_data(), host_grid.get_data(), device_grid.byte_size(), hipMemcpyHostToDevice);

    }
    {
        // ensure c-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellActive,  fake_cuda_allocator<char>, true> device_grid(20, 10, 5);
        BOOST_TEST(data == device_grid.get_data());

        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        hipMemcpy(host_grid.get_data(), device_grid.get_data(), device_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = host_grid.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 100000;

                    BOOST_TEST(cell.element.val == expected);
                    BOOST_TEST(cell.temperature == 0);
                    BOOST_TEST(cell.alive == false);
                }
            }
        }
    }
    {
        // ensure d-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        hipMemcpy(host_grid.get_data(), data, host_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = host_grid.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 1100000;

                    BOOST_TEST(cell.element.val == expected);
                    BOOST_TEST(cell.temperature == 0);
                    BOOST_TEST(cell.alive == false);
                }
            }
        }
    }

    fake_cuda_allocator<char>().deallocate_all();
}

ADD_TEST(TestCUDAGetSetSingleElements)
{
    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(40, 13, 8);

    for (int z = 0; z < 8; ++z) {
        for (int y = 0; y < 13; ++y) {
            for (int x = 0; x < 40; ++x) {
                ConstructorDestructorTestCellPassive cell((x + 2) * (y + 2), true);
                cell.element.val = 10000 + x + y * 40 + z * 40 * 13;
                device_grid.set(x, y, z, cell);
            }
        }
    }

    for (int z = 0; z < 8; ++z) {
        for (int y = 0; y < 13; ++y) {
            for (int x = 0; x < 40; ++x) {
                ConstructorDestructorTestCellPassive cell = device_grid.get(x, y, z);

                int expected = 10000 + x + y * 40 + z * 40 * 13;
                BOOST_TEST(cell.element.val == expected);
                BOOST_TEST(cell.temperature == ((x + 2) * (y + 2)));
                BOOST_TEST(cell.alive       == true);
            }
        }
    }
}

ADD_TEST(TestCUDAGetSetMultipleElements)
{
    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(35, 25, 15);

    for (int z = 0; z < 15; ++z) {
        for (int y = 0; y < 25; ++y) {
            std::vector<ConstructorDestructorTestCellPassive> cells(35);
            for (int x = 0; x < 35; ++x) {
                cells[x].alive = x % 2;
                cells[x].temperature = x * y * z;
                cells[x].element.val = 20000 + x + y * 35 + z * 35 * 25;
            }

            device_grid.set(0, y, z, cells.data(), 35);
        }
    }

    for (int z = 0; z < 15; ++z) {
        for (int y = 0; y < 25; ++y) {
            std::vector<ConstructorDestructorTestCellPassive> cells(35);
            device_grid.get(0, y, z, cells.data(), 35);

            for (int x = 0; x < 35; ++x) {
                int expected = 20000 + x + y * 35 + z * 35 * 25;

                BOOST_TEST(cells[x].element.val == expected);
                BOOST_TEST(cells[x].alive == (x % 2));
                BOOST_TEST(cells[x].temperature == (x * y * z));
            }
        }
    }
}

ADD_TEST(TestCUDALoadSaveElements)
{
    soa_grid<ConstructorDestructorTestCellPassive> host_grid(21, 10, 9);
    for (int z = 0; z < 9; ++z) {
        for (int y = 0; y < 10; ++y) {
            for (int x = 0; x < 21; ++x) {
                ConstructorDestructorTestCellPassive cell;
                cell.alive = ((x % 3) == 0);
                cell.temperature = x * y * z * -1;
                cell.element.val = 30000 + x + y * 21 + z * 21 * 10;
                host_grid.set(x, y, z, cell);
            }
        }
    }

    std::vector<char> buffer(10 * aggregated_member_size<ConstructorDestructorTestCellPassive>::VALUE);
    host_grid.save(11, 9, 8, buffer.data(), 10);

    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(31, 20, 19);
    device_grid.load(21, 19, 18, buffer.data(), 10);

    for (int i = 0; i < 20; ++i) {
        ConstructorDestructorTestCellPassive cell;
        cell.alive = i % 4;
        cell.temperature = 4711 + i;
        cell.element.val = 100 * i;
        device_grid.set(i + 1, 5, 6, cell);
    }

    buffer.resize(20 * aggregated_member_size<ConstructorDestructorTestCellPassive>::VALUE);
    device_grid.save(1, 5, 6, buffer.data(), 20);

    // very load:
    soa_grid<ConstructorDestructorTestCellPassive> host_grid2(31, 20, 19);
    hipMemcpy(host_grid2.get_data(), device_grid.get_data(), device_grid.byte_size(), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        ConstructorDestructorTestCellPassive cell = host_grid2.get(21 + i, 19, 18);

        bool expectedAlive = (((i + 11) % 3) == 0);
        double expectedTemperature = (11 + i) * 9 * 8 * -1;
        int expectedVal = 30000 + (11 + i) + 9 * 21 + 8 * 21 * 10;

        BOOST_TEST(cell.alive == expectedAlive);
        BOOST_TEST(cell.temperature == expectedTemperature);
        BOOST_TEST(cell.element.val == expectedVal);
    }

    // verify save:
    double *temperature = (double*)(buffer.data() +  0 * 20);
    int *val            = (int*)   (buffer.data() +  8 * 20);
    bool *alive         = (bool*)  (buffer.data() + 12 * 20);

    for (int i = 0; i < 20; ++i) {
        bool expectedAlive = i % 4;
        double expectedTemperature = 4711 + i;
        int expectedVal = i * 100;

        BOOST_TEST(expectedAlive       == alive[i]);
        BOOST_TEST(expectedTemperature == temperature[i]);
        BOOST_TEST(expectedVal         == val[i]);
    }

    // sanity check:
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(error) << "\n";
        throw std::runtime_error("CUDA error");
    }
}

// fixme: need test with array member, too!
}

int main(int argc, char **argv)
{
    return 0;
}
